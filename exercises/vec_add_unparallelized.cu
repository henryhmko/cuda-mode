
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// UNPARALLELIZED VERSION

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y) {
    for (int i = 0; i < n; i++) {
        y[i] = x[i] + y[i];
    }
}

int main(void) {
    // Left-shift operator used to set N to 1*2^20, which is 1,048,576(approx 1mil)
    int N = 1 << 20;
    float *x, *y;

    // Allocate Unified Memory -- accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // Initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    
    // Run kernel on 1M elements on the GPU
    add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    printf("Max error: %f\n", maxError);

    // Free memory
    hipFree(x);
    hipFree(y);
    
    return 0; 
}


// Additional steps for Profiling:
// $ nvprof ./vec_add_unparallelized

// Time: 52.624ms on RTX 2080ti
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

extern "C" {
int main() {
    // define M, N, K, alpha, beta
    int M = 4092, N = 4092, K = 4092;
    float alpha = 1.0f, beta = 0.0f;

    // allocate and initialize host matrices
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    
    // initialize
    std::fill_n(h_A, M * K, 2.0f);
    std::fill_n(h_B, K * N, 1.0f);
    std::fill_n(h_C, M * N, 0.0f);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // move data from host to device
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice);

    // create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // create profiling for kernel 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start timing
    hipEventRecord(start);

    // perform matrix multiplication using cuBLAS
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, 
                &alpha, d_B, N, d_A, K, &beta, d_C, N);

    // stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // copy result back to host
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // get elapsed time
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "cuBLAS SGEMM execution time: " << ms << " ms" << std::endl;

    // destroy cuda events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // destroy cuBLAS handle
    hipblasDestroy(handle);

    // free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
}